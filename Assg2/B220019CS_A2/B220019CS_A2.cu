#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <fstream>

#define CUDA_CHK(...) { \
  hipError_t cuda_err_code = __VA_ARGS__; \
  if (cuda_err_code != hipSuccess) { \
    printf("%s failed with code %d\n", #__VA_ARGS__, cuda_err_code); \
    abort(); \
  } \
}

#define TINY 1.0e-40
#define a(i,j,N) a[(i)*(N)+(j)]

// Function to print matrices to a file in row-major order
void print_output(FILE* output_file, int N, float* L, float* U, float* X) {
    fprintf(output_file, "%d\n", N); // First line is N

    // Printing the lower triangular matrix L
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (i > j) {
                fprintf(output_file, "%f\n", L[i * N + j]);
            } else if (i == j) {
                fprintf(output_file, "1.000000\n");  // Diagonal is 1 in L
            } else {
                fprintf(output_file, "0.000000\n");  // Upper part is 0
            }
        }
        //fprintf(output_file, "\n");
    }

    // Printing the upper triangular matrix U
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (i <= j) {
                fprintf(output_file, "%f\n", U[i * N + j]);
            } else {
                fprintf(output_file, "0.000000\n");  // Lower part is 0
            }
        }
        //fprintf(output_file, "\n");
    }

    // Printing the solution vector X
    for (int i = 0; i < N; i++) {
        fprintf(output_file, "%f\n", X[i]);
    }
}

__device__ void d_pivot_decomp(float *a, int *p, int *q, int N, float *L, float *U){
    int i,j,k;
    int pi,pj,tmp;
    float max;
    float ftmp;

    // Initialize U with values of A and L as 0
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            U[i * N + j] = a[i * N + j];
            L[i * N + j] = (i == j) ? 1.0f : 0.0f;  // Initialize diagonal of L to 1
        }
    }

    for (k=0;k<N;k++){
        pi=-1,pj=-1,max=0.0;
        for (i=k;i<N;i++) {
            for (j=k;j<N;j++) {
                if (fabs(a(i,j,N))>max){
                    max = fabs(a(i,j,N));
                    pi=i;
                    pj=j;
                }
            }
        }
        tmp=p[k];
        p[k]=p[pi];
        p[pi]=tmp;
        for (j=0;j<N;j++){
            ftmp=a(k,j,N);
            a(k,j,N)=a(pi,j,N);
            a(pi,j,N)=ftmp;
        }
        tmp=q[k];
        q[k]=q[pj];
        q[pj]=tmp;
        for (i=0;i<N;i++){
            ftmp=a(i,k,N);
            a(i,k,N)=a(i,pj,N);
            a(i,pj,N)=ftmp;
        }
        if ((fabs(a(k,k,N))>TINY)){
            for (i=k+1;i<N;i++){
                ftmp=a(i,k,N)/=a(k,k,N);
                L[i * N + k] = ftmp;  // Store lower triangular values in L
                for (j=k+1;j<N;j++){
                    a(i,j,N)-=(ftmp*a(k,j,N));
                    U[i * N + j] = a(i,j,N);  // Store upper triangular values in U
                }
            }
        }
    }
}

__device__ void d_solve(float *a, float *x, int *p, int *q, int N){
    int i, ii = 0, j;
    float ftmp;
    float *xtmp = new float[N];  
    int *inverse_q = new int[N];  

    for (i = 0; i < N; i++) {
        inverse_q[q[i]] = i;
    }

    for (i = 0; i < N; i++) {
        xtmp[i] = x[p[i]]; 
    }

    for (i = 0; i < N; i++) {
        ftmp = xtmp[i];
        if (ii != 0)
            for (j = ii - 1; j < i; j++)
                ftmp -= a(i,j,N) * xtmp[j];
        else if (ftmp != 0.0)
            ii = i + 1;
        xtmp[i] = ftmp;
    }

    xtmp[N - 1] /= a(N-1, N-1, N);
    for (i = N - 2; i >= 0; i--) {
        ftmp = xtmp[i];
        for (j = i + 1; j < N; j++) {
            ftmp -= a(i,j,N) * xtmp[j];
        }
        xtmp[i] = ftmp / a(i,i,N);
    }

    for (i = 0; i < N; i++) {
        x[i] = xtmp[inverse_q[i]];
    }

    delete[] xtmp;    
    delete[] inverse_q;  
}

__global__ void solve(float *A, float *B, int max, int N, float *L, float *U){
  int id = blockDim.x*blockIdx.x + threadIdx.x;
  extern __shared__ float shared_mem[];  // Shared memory
  
  int *p_pivot = new int[N];  
  int *q_pivot = new int[N];

  float *local_A = shared_mem;  // Shared memory for matrix A
  float *local_B = &shared_mem[N * N];  // Shared memory for vector B
  
  if ((id < max)){
    for (int i = 0; i < N; i++) {
        p_pivot[i] = q_pivot[i] = i;
    }

    for (int i = 0; i < N * N; i++) {
        local_A[i] = A[id * N * N + i];  // Load data into shared memory
    }

    for (int i = 0; i < N; i++) {
        local_B[i] = B[id * N + i];
    }

    d_pivot_decomp(local_A, p_pivot, q_pivot, N, L, U);
    d_solve(local_A, local_B, p_pivot, q_pivot, N);

    for (int i = 0; i < N; i++) {
        B[id * N + i] = local_B[i];  // Write back the result
    }
  }

  delete[] p_pivot;  
  delete[] q_pivot;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {  // Third argument for timing output file
        fprintf(stderr, "Usage: %s <inputfile> <outputfile> <timingfile>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char *input_filename = argv[1];
    const char *output_filename = argv[2];
    const char *timing_filename = argv[3];

    int N;
    float *a, *b;

    // Using <chrono> for timing
    using namespace std::chrono;
    high_resolution_clock::time_point start, end;
    duration<double> time_taken;

    // Timing for reading matrices
    start = high_resolution_clock::now();

    FILE *file = fopen(input_filename, "r");
    if (file == NULL) {
        fprintf(stderr, "Error opening file %s.\n", input_filename);
        return EXIT_FAILURE;
    }

    // fscanf(file, "%d", &N);

    if (fscanf(file, "%d", &N) != 1) {
        fprintf(stderr, "Error reading N from file.\n");
        return EXIT_FAILURE;
    }

    a = (float *)malloc(N * N * sizeof(float));
    b = (float *)malloc(N * sizeof(float));

    // for (int i = 0; i < N; i++)
    //     for (int j = 0; j < N; j++)
    //         fscanf(file, "%f", &a[i * N + j]);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (fscanf(file, "%f", &a[i * N + j]) != 1) {
                fprintf(stderr, "Error reading matrix A from file.\n");
                return EXIT_FAILURE;
            }
        }
    }

    for (int i = 0; i < N; i++) {
        if (fscanf(file, "%f", &b[i]) != 1) {
            fprintf(stderr, "Error reading vector B from file.\n");
            return EXIT_FAILURE;
        }
    }

    
    fclose(file);

    end = high_resolution_clock::now();
    time_taken = duration_cast<duration<double>>(end - start);

    // Open timing file
    std::ofstream timing_file(timing_filename);
    if (!timing_file) {
        std::cerr << "Error opening timing file " << timing_filename << std::endl;
        return EXIT_FAILURE;
    }

    timing_file << "Time taken to read A and B: " << time_taken.count() << " seconds\n";

    // CUDA events for timing GPU computations
    hipEvent_t cuda_start, cuda_stop;
    float milliseconds = 0;

    hipSetDevice(0);
    float* d_A;
    float* d_b;
    float* d_L;
    float* d_U;

    CUDA_CHK(hipMalloc((void**)&d_A, sizeof(float) * N * N));
    CUDA_CHK(hipMalloc((void**)&d_b, sizeof(float) * N));
    CUDA_CHK(hipMalloc((void**)&d_L, sizeof(float) * N * N));
    CUDA_CHK(hipMalloc((void**)&d_U, sizeof(float) * N * N));

    CUDA_CHK(hipMemcpy(d_A, a, sizeof(float) * N * N, hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

    // Start timing for L and U computation
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_stop);
    hipEventRecord(cuda_start);

    int shared_size = (N * N + N) * sizeof(float);  // Memory for shared A and B
    int M = 100;  // Assume we have 10 systems to solve in parallel
    int threadsPerBlock = N;  // Number of threads per block, each handling one row
    int blocksPerGrid = (M + threadsPerBlock - 1) / threadsPerBlock;  // Number of blocks
    solve<<<blocksPerGrid, threadsPerBlock, shared_size>>>(d_A, d_b, 1, N, d_L, d_U);  // Kernel with shared memory

    hipEventRecord(cuda_stop);
    hipEventSynchronize(cuda_stop);
    hipEventElapsedTime(&milliseconds, cuda_start, cuda_stop);
    timing_file << "Time taken in computing L and U: " << milliseconds / 1000 << " seconds\n";

    CUDA_CHK(hipMemcpy(b, d_b, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Allocate space for L and U matrices
    float* L = (float*)malloc(N * N * sizeof(float));
    float* U = (float*)malloc(N * N * sizeof(float));
    CUDA_CHK(hipMemcpy(L, d_L, sizeof(float) * N * N, hipMemcpyDeviceToHost));
    CUDA_CHK(hipMemcpy(U, d_U, sizeof(float) * N * N, hipMemcpyDeviceToHost));

    FILE *output_file = fopen(output_filename, "w");
    if (output_file == NULL) {
        fprintf(stderr, "Error opening output file %s.\n", output_filename);
        return EXIT_FAILURE;
    }

    print_output(output_file, N, L, U, b);
    fclose(output_file);

    // Report total time taken to solve system
    timing_file << "Total time taken to solve system: " << milliseconds / 1000 << " seconds\n";

    // Clean up
    timing_file.close();

    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_L);
    hipFree(d_U);
    free(a);
    free(b);
    free(L);
    free(U);

    return 0;
}
