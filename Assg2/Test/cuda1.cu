
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA() {
    printf("Hello, CUDA!\n");
}

int main() {
    helloCUDA<<<1,1>>>();
    printf("Hello, CUDAaaaaaa!\n");
    hipDeviceSynchronize();
    return 0;
}